#include "hip/hip_runtime.h"
#include "VX3_VoxelyzeKernel.h"
#include "VX3_MemoryCleaner.h"

/* Sub GPU Threads */
__global__ void gpu_update_force(TI_Link* links, int num);
__global__ void gpu_update_voxel(TI_Voxel* voxels, int num, double dt);
__global__ void gpu_update_temperature(TI_Voxel* voxels, int num, double currentTemperature);

/* Host methods */

VX3_VoxelyzeKernel::VX3_VoxelyzeKernel(CVoxelyze* In, hipStream_t In_stream)
{
    stream = In_stream;

    voxSize = In->voxSize;
    
    num_d_linkMats = In->linkMats.size();
    VcudaMalloc( (void **)&d_linkMats, num_d_linkMats * sizeof(TI_MaterialLink));
    {
        int i = 0;
        for (auto mat:In->linkMats) {
            TI_MaterialLink tmp_linkMat( mat, stream );
            VcudaMemcpyAsync( d_linkMats+i, &tmp_linkMat, sizeof(TI_MaterialLink), VcudaMemcpyHostToDevice, stream );
            h_linkMats.push_back( mat );
            i++;
        }
    }

    num_d_voxels = In->voxelsList.size();
    VcudaMalloc( (void **)&d_voxels, num_d_voxels * sizeof(TI_Voxel));
    for (int i=0;i<num_d_voxels;i++) {
        h_voxels.push_back( In->voxelsList[i] );
    }

    num_d_links = In->linksList.size();
    VcudaMalloc( (void **)&d_links, num_d_links * sizeof(TI_Link));
    for (int i=0;i<num_d_links;i++) {
        TI_Link tmp_link( In->linksList[i], this );
        VcudaMemcpyAsync( d_links+i, &tmp_link, sizeof(TI_Link), VcudaMemcpyHostToDevice, stream );
        h_links.push_back( In->linksList[i] );
    }

    for (int i=0;i<num_d_voxels;i++) {
        //set values for GPU memory space
        TI_Voxel tmp_voxel(In->voxelsList[i], this);
        VcudaMemcpyAsync(d_voxels+i, &tmp_voxel, sizeof(TI_Voxel), VcudaMemcpyHostToDevice, stream);
    }

    // VcudaMalloc((void**)&d_collisionsStale, sizeof(bool));

    // VcudaMalloc((void **)&d_collisions, sizeof(TI_vector<TI_Collision *>));
    // VcudaMemcpyAsync(d_collisions, &h_collisions, sizeof(TI_vector<TI_Collision *>), VcudaMemcpyHostToDevice, stream);

}

void VX3_VoxelyzeKernel::cleanup() {
    //The reason not use ~VX3_VoxelyzeKernel is that will be automatically call multiple times after we use memcpy to clone objects.
    MycudaFree(d_linkMats);
    MycudaFree(d_voxels);
    MycudaFree(d_links);
    MycudaFree(d_collisionsStale);
    // MycudaFree(d_collisions);
}

TI_MaterialLink * VX3_VoxelyzeKernel::getMaterialLink(CVX_MaterialLink* vx_mats) {
    for (int i=0;i<num_d_linkMats;i++) {
        if (h_linkMats[i] == vx_mats) {
            return &d_linkMats[i];
        }
    }
    printf("ERROR: Cannot find the right link material. h_linkMats.size() %ld.\n", h_linkMats.size());
    return NULL;
}

/* Cuda methods : cannot use any CVX_xxx, and no std::, no boost::, and no filesystem. */

__device__ void VX3_VoxelyzeKernel::syncVectors() {
    for (int i=0;i<num_d_linkMats;i++) {
        d_linkMats[i].syncVectors();
    }
}
__device__ bool VX3_VoxelyzeKernel::StopConditionMet(void) //have we met the stop condition yet?
{
    if (StopConditionType!=SC_MAX_SIM_TIME) {
        printf(COLORCODE_BOLD_RED "Only support this type of stop condition for now.\n" COLORCODE_RESET);
        return true;
    }
    return currentTime > StopConditionValue ? true : false;
}

__device__ double VX3_VoxelyzeKernel::recommendedTimeStep() {
    //find the largest natural frequency (sqrt(k/m)) that anything in the simulation will experience, then multiply by 2*pi and invert to get the optimally largest timestep that should retain stability
	double MaxFreq2 = 0.0f; //maximum frequency in the simulation in rad/sec
    for (int i=0;i<num_d_links;i++) {
        TI_Link* pL = d_links+i;
		//axial
		double m1 = pL->pVNeg->mat->mass(),  m2 = pL->pVPos->mat->mass();
		double thisMaxFreq2 = pL->axialStiffness()/(m1<m2?m1:m2);
		if (thisMaxFreq2 > MaxFreq2) MaxFreq2 = thisMaxFreq2;
		//rotational will always be less than or equal
	}
	if (MaxFreq2 <= 0.0f){ //didn't find anything (i.e no links) check for individual voxelss
		for (int i=0;i<num_d_voxels;i++){ //for each link
			double thisMaxFreq2 = d_voxels[i].mat->youngsModulus() * d_voxels[i].mat->nomSize / d_voxels[i].mat->mass(); 
			if (thisMaxFreq2 > MaxFreq2) MaxFreq2 = thisMaxFreq2;
		}
	}
	if (MaxFreq2 <= 0.0f) return 0.0f;
	else return 1.0f/(6.283185f*sqrt(MaxFreq2)); //the optimal timestep is to advance one radian of the highest natural frequency
}

__device__ void VX3_VoxelyzeKernel::updateTemperature() {
    //updates the temperatures For Actuation!
    // different temperatures in different objs are not support for now.
    if (VaryTempEnabled){
		if (TempPeriod > 0) {
            currentTemperature = TempBase + TempAmplitude*sin(2*3.1415926/TempPeriod* currentTime);	//update the global temperature
            int blockSize = 512;
            int gridSize_voxels = (num_d_voxels + blockSize - 1) / blockSize; 
            int blockSize_voxels = num_d_voxels<blockSize ? num_d_voxels : blockSize;
            gpu_update_temperature<<<gridSize_voxels, blockSize_voxels>>>(d_voxels, num_d_voxels, currentTemperature - TempBase);
            hipDeviceSynchronize();        
        }
	}
}

__device__ bool VX3_VoxelyzeKernel::doTimeStep(float dt) {
    updateTemperature();
    CurStepCount++;
	if (dt==0) return true;
	else if (dt<0) {
        if (!OptimalDt) {
            OptimalDt = recommendedTimeStep();
        }
        if (OptimalDt<1e-10) {
            CUDA_DEBUG_LINE("recommendedTimeStep is zero.");
            return false;
        }
        dt = DtFrac*OptimalDt;
    }
    bool Diverged = false;

    int blockSize = 512;
    int gridSize_links = (num_d_links + blockSize - 1) / blockSize; 
    int blockSize_links = num_d_links<blockSize ? num_d_links : blockSize;
    gpu_update_force<<<gridSize_links, blockSize_links>>>(d_links, num_d_links);
    hipDeviceSynchronize();

    for (int i = 0; i<num_d_links; i++){
        if (d_links[i].axialStrain() > 100){
            CUDA_DEBUG_LINE("Diverged.");
            Diverged = true; //catch divergent condition! (if any thread sets true we will fail, so don't need mutex...
        }
    }
    if (Diverged) return false;

    // 	if (collisions) updateCollisions();

    int gridSize_voxels = (num_d_voxels + blockSize - 1) / blockSize; 
    int blockSize_voxels = num_d_voxels<blockSize ? num_d_voxels : blockSize;
    gpu_update_voxel<<<gridSize_voxels, blockSize_voxels>>>(d_voxels, num_d_voxels, dt);
    hipDeviceSynchronize();

    currentTime += dt;
    return true;
}

__device__ void VX3_VoxelyzeKernel::updateCurrentCenterOfMass() {
	double TotalMass = 0;
	TI_Vec3D<> Sum(0,0,0);
	for (int i=0; i<num_d_voxels; i++){
        double ThisMass = d_voxels[i].material()->mass();
		Sum += d_voxels[i].position()*ThisMass;
        TotalMass += ThisMass;
	}

	currentCenterOfMass = Sum/TotalMass;
}

/* Sub GPU Threads */
__global__ void gpu_update_force(TI_Link* links, int num) {
    int gindex = threadIdx.x + blockIdx.x * blockDim.x;
    if (gindex < num) {
        TI_Link* t = &links[gindex];
        t->updateForces();
        if (t->axialStrain() > 100) { printf("ERROR: Diverged."); }
    }
}
__global__ void gpu_update_voxel(TI_Voxel* voxels, int num, double dt) {
    int gindex = threadIdx.x + blockIdx.x * blockDim.x; 
    if (gindex < num) {
        TI_Voxel* t = &voxels[gindex];
        t->timeStep(dt);
    }
}

__global__ void gpu_update_temperature(TI_Voxel* voxels, int num, double temperature) {
    int gindex = threadIdx.x + blockIdx.x * blockDim.x; 
    if (gindex < num) {
        TI_Voxel* t = &voxels[gindex];
        t->setTemperature(temperature);
    }
}