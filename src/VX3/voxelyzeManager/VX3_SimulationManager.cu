#include "hip/hip_runtime.h"
#include "VX3_SimulationManager.h"
#include "VX3_VoxelyzeKernel.h"
#include "VX_Sim.h"


__global__ void CUDA_Simulation(VX3_VoxelyzeKernel *d_voxelyze_3, int num_tasks) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i<num_tasks) {
        VX3_VoxelyzeKernel *d_v3 = &d_voxelyze_3[i];
        if (i==1) {
            printf("debug.\n");
        }
        printf("--> simulation %d runs.\n", i);
        for (int j=0;j<1000000;j++) { //Maximum Steps 1000000
            if (d_v3->StopConditionMet()) break;
            // if (j%1000==0) {
            //     printf("----> [Task %d] doTimeStep %d, Current Time (in sec) %f \t", i, j, d_v3->currentTime);
            //     d_v3->updateCurrentCenterOfMass();
            //     printf("Current Location (in meter): %f %f %f\n", d_v3->currentCenterOfMass.x, d_v3->currentCenterOfMass.y, d_v3->currentCenterOfMass.z);
            // }
            if (!d_v3->doTimeStep()) {
                printf("--> simulation %d Diverged.\n", i);
                break;
            }
        }
        d_v3->updateCurrentCenterOfMass();
        printf("--> simulation %d ends.\n", i);
    }
}

void VX3_SimulationManager::operator()(VX3_TaskManager* tm, fs::path batchFolder) {
    //TODO: 
    //1. read every VXA files
    VX3_VoxelyzeKernel * d_voxelyze_3;
    std::vector<VX3_VoxelyzeKernel *> h_d_voxelyze_3;
    int batch_size = 0;
    for (auto &file : fs::directory_iterator( batchFolder) ) { batch_size++; }
    hipMalloc((void**)&d_voxelyze_3, batch_size * sizeof(VX3_VoxelyzeKernel));

    int i = 0;
    for (auto &file : fs::directory_iterator( batchFolder ) ) {
        CVX_Environment MainEnv;
        CVX_Sim MainSim;
        CVX_Object MainObj;
        MainEnv.pObj = &MainObj; //connect environment to object
        MainSim.pEnv = &MainEnv; //connect Simulation to envirnment
        MainSim.LoadVXAFile(file.path().string());
        std::string err_string; //need to link this up to get info back...
        if (!MainSim.Import(NULL, NULL, &err_string)){
            std::cout<<err_string;
        }
        VX3_VoxelyzeKernel h_d_tmp(&MainSim.Vx);
        h_d_tmp.DtFrac = MainSim.DtFrac;
        h_d_tmp.StopConditionType = MainSim.StopConditionType;
        h_d_tmp.StopConditionValue = MainSim.StopConditionValue;
        h_d_tmp.TempEnabled = MainSim.pEnv->TempEnabled;
        h_d_tmp.VaryTempEnabled = MainSim.pEnv->VaryTempEnabled;
        h_d_tmp.TempBase = MainSim.pEnv->TempBase;
        h_d_tmp.TempAmplitude = MainSim.pEnv->TempAmplitude;
        h_d_tmp.TempPeriod = MainSim.pEnv->TempPeriod;
        h_d_tmp.currentTemperature = h_d_tmp.TempBase + h_d_tmp.TempAmplitude;
        
        hipMemcpy(d_voxelyze_3 + i, &h_d_tmp, sizeof(VX3_VoxelyzeKernel), hipMemcpyHostToDevice);
        i++;
    }

    //3. start CUDA Simulation
    int num_tasks = batch_size;
    int threadsPerBlock = 512;
    int numBlocks = (num_tasks + threadsPerBlock - 1) / threadsPerBlock;
    if (numBlocks == 1)
        threadsPerBlock = num_tasks;
    CUDA_Simulation<<<numBlocks,threadsPerBlock>>>(d_voxelyze_3, num_tasks);

    //4. wait
    hipDeviceSynchronize();
    
    //5. read result
    double final_z = 0.0;
    VX3_VoxelyzeKernel* result_voxelyze_kernel = (VX3_VoxelyzeKernel *)malloc(num_tasks * sizeof(VX3_VoxelyzeKernel));
    hipMemcpy( result_voxelyze_kernel, d_voxelyze_3, num_tasks * sizeof(VX3_VoxelyzeKernel), hipMemcpyDeviceToHost );
    //TODO: how to communicate with experiments? files? or other methods?
    printf("====[RESULTS for %s]====", batchFolder.filename());
    for (int i=0;i<num_tasks;i++) {
        auto ret = &result_voxelyze_kernel[i];
        printf("Task %d: position (in mm): %f %f %f, end time: %f\n", i,
        ret->currentCenterOfMass.x*1000, ret->currentCenterOfMass.y*1000, ret->currentCenterOfMass.z*1000,
        ret->currentTime
        );
    }
    //6. cleanup
    for (auto p:h_d_voxelyze_3) {
        p->cleanup();
    }
    hipFree(d_voxelyze_3);
    tm->cleanBatchFolder(batchFolder);
    // delete result_voxelyze_kernel;

    return;
}